#include "hip/hip_runtime.h"
/*
 * main.cpp
 *
 *  Created on: Sep 11, 2014
 *      Author: Abuenameh
 */

#include <ctime>

#include "cusimann.cuh"
#include "nelderMead.h"
#include "gutzwiller.hpp"
//#include "energy.cuh"


int main(int argc, char** argv) {
	time_t start = time(NULL);

	real T_0 = 1000, T_min = 0.01;
	const unsigned int n = 2 * L * dim, N = 10;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*) malloc(sizeFD),
			f_cusimann_minimum;
	lb = (real*) malloc(sizeFD);
	unsigned int i;
	for (i = 0; i < n; i++)
		lb[i] = -1;
	ub = (real*) malloc(sizeFD);
	for (i = 0; i < n; i++)
		ub[i] = 1;

	unsigned int n_threads_per_block = 128;//512;//256;
	unsigned int n_blocks = 64;

	real U[L], J[L];
	for (int i = 0; i < L; i++) {
		U[i] = 1;
		J[i] = 0.001;
	}
	parameters parms;
	parms.U = U;
	parms.J = J;
	parms.mu = 0.5;

	parameters* d_parms;
	real* d_U;
	real* d_J;
	checkCudaErrors(hipMalloc(&d_U, L*sizeof(real)));
	checkCudaErrors(hipMemcpy(d_U, U, L*sizeof(real), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_J, L*sizeof(real)));
	checkCudaErrors(hipMemcpy(d_J, J, L*sizeof(real), hipMemcpyHostToDevice));

	real theta = 0;

	parms.U = d_U;
	parms.J = d_J;
	parms.mu = 0.5;
	parms.theta = theta;
	parms.costh = cos(theta);
	parms.sinth = sin(theta);
	parms.cos2th = cos(2*theta);
	parms.sin2th = sin(2*theta);
	checkCudaErrors(hipMalloc(&d_parms, sizeof(parameters)));
	checkCudaErrors(
			hipMemcpy(d_parms, &parms, sizeof(parameters),
					hipMemcpyHostToDevice));

	printf("Optimize\n");
	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb,
			ub, Energy<real>(), d_parms, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	parms.U = U;
	parms.J = J;

	double f_nelderMead_minimum;
	double *nelderMead_minimum = (double*) malloc(n * sizeof(double));
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, &parms,
			nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);

	time_t end = time(NULL);

	printf("Runtime: %ld s\n", end-start);

	return 0;
}

template<class T>
class Ackley {
public:
	__host__ __device__ T operator()(const T *x, unsigned int n,
			void *f_data) const {
		T f_x = 0.0f;
		T aux = exp(-0.2f);

		int i;
		for (i = 0; i < n - 1; i++)
			f_x = f_x + aux * sqrt(pow(x[i], 2) + pow(x[i + 1], 2))
					+ 3.0f * (cos(2.0f * x[i]) + sin(2.0f * x[i + 1]));

		printf("Iter\n");
		return f_x;
	}
};

double f_nelderMead2(unsigned int n, const double *x, double *grad,
		void *f_data) {
	return Ackley<double>()(x, n, f_data);
}

int main2() {
	real T_0 = 1000, T_min = 0.1;
	const unsigned int n = 5, N = 100;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*) malloc(sizeFD),
			f_cusimann_minimum;
	lb = (real*) malloc(sizeFD);
	unsigned int i;
	for (i = 0; i < n; i++)
		lb[i] = -30;
	ub = (real*) malloc(sizeFD);
	for (i = 0; i < n; i++)
		ub[i] = 30;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb,
			ub, Ackley<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	hipDeviceSynchronize();
	printf("cusimann_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double f_nelderMead_minimum;
	double *nelderMead_minimum = (double*) malloc(n * sizeof(double));
//	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead2, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for (i = 0; i < n; i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");


	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);

	return EXIT_SUCCESS;
}
